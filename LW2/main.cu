#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <ctime>
#include <algorithm>

#include <D_Matrix.cuh>
#include <H_Matrix.cuh>
#include <chronoCPU.hpp>
#include <chronoGPU.hpp>

#include <hip/hip_runtime_api.h>
#include <helper_string.h>

namespace {

	void displayHelpIfNeeded(const int argc, const char**argv) 
	{
		if( checkCmdLineFlag(argc, argv, "-h") || checkCmdLineFlag(argc, argv, "help") ) {
			std::cout << "Usage: " << argv[0] << " [-h] [--help] [-n=xxx]" << std::endl;
			std::cout << "\twhere options -h and --help display this help," << std::endl;
			std::cout << "\t  and option -n=xxx sets the width of the matrices to xxx" << std::endl;
			exit(0);
		}
	}

	int getNFromCmdLine(const int argc, const char**argv) 
	{
		const int minN = 1<<3;
		const int maxN = 1<<14;
		int N = 1<<8;
		if( checkCmdLineFlag(argc, argv, "n") ) {
			int value;
			getCmdLineArgumentValue(argc, argv, "n", &value);
			std::cout << "\tfind command line parameter -n=" << value << std::endl;
			if( value >= minN && value <= maxN )
				N = value;
			else
				std::cerr << "\tWarning: parameter must be greater to " << minN << " and lesser than " << maxN << std::endl;
		}
		return N;
	}

	void matrix_print(const H_Matrix& M) {
		for (int l = 0; l < M.m_n; ++l)
		{
			for (int c = 0; c < M.m_n; ++c)
				printf("% 4d ", M.h_val[l*M.m_n + c]);
			putchar('\n');
		}
	}

	bool test_equality(const H_Matrix& hM) 
	{
		return hM == hM;
	}

	bool test_addition(const H_Matrix& hA, const H_Matrix&hB, const D_Matrix& dA, const D_Matrix&dB) {
		// do the addition on GPU
		ChronoGPU gChr;
		gChr.start();
		const D_Matrix dResult = dA + dB;
		gChr.stop();

		// ground truth (CPU)
		ChronoCPU cChr;
		cChr.start();
		H_Matrix truth = hA + hB;
		cChr.stop();

		// print the timings
		std::cout << "==========================================" << std::endl;
		std::cout << "-> Matrices addition ..." << std::endl;
		std::cout << "Computation time on GPU:" << gChr.elapsedTime() << "ms." << std::endl;
		std::cout << "Computation time on CPU:" << cChr.elapsedTimeInMilliSeconds() << "ms." << std::endl;
		std::cout << "==========================================" << std::endl;

		H_Matrix hResult(hA.m_n);
		dResult.data(hResult.h_val);

		if (hA.m_n < 10) {
			std::cout << "Truth" << std::endl;
			matrix_print(truth);
			std::cout << "Device" << std::endl;
			matrix_print(hResult);
		}
		return truth == hResult;
	}

	bool test_transpose(const H_Matrix& hM, const D_Matrix& dM) 
	{
		// do the transpose on GPU
		ChronoGPU gChr;
		gChr.start();
		const D_Matrix dResult = dM.transpose();
		gChr.stop();

		// ground truth (CPU)
		ChronoCPU cChr;
		cChr.start();
		H_Matrix truth = hM.transpose();
		cChr.stop();

		// print the timings
		std::cout << "==========================================" << std::endl;
		std::cout << "-> Matrix transpose ..." << std::endl;
		std::cout << "Computation time on GPU:" << gChr.elapsedTime() << "ms." << std::endl;
		std::cout << "Computation time on CPU:" << cChr.elapsedTimeInMilliSeconds() << "ms." << std::endl;
		std::cout << "==========================================" << std::endl;

		H_Matrix hResult(hM.m_n);
		dResult.data(hResult.h_val);
		return truth == hResult;
	}


	bool test_diffusion(const H_Matrix& hA, const D_Matrix& dA)
	{
		// allocate the result
		D_Matrix diffused(hA.m_n);
		H_Matrix result(hA.m_n);
		H_Matrix truth(hA.m_n);
		// do it
		ChronoGPU gChr;
		ChronoCPU cChr;
		float gElapsed = 0.f, cElapsed = 0.f;
		int nb;
		const int nbLoops = (std::min)(128, hA.m_n);
		for (nb = 0; nb < nbLoops; ++nb) {
			gChr.start();
			dA.diffusion(nb, diffused);
			gChr.stop();
			gElapsed += gChr.elapsedTime();
			diffused.data(result.h_val);
			cChr.start();
			hA.diffusion(nb, truth);
			cChr.stop();
			cElapsed += cChr.elapsedTimeInMilliSeconds();
			if (result != truth)
				break;
		}	
		// print the timings
		std::cout << "==========================================" << std::endl;
		std::cout << "-> Diffusion of matrix (" << nb << " loops) ..." << std::endl;
		std::cout << "Computation time on GPU:" << (gElapsed/static_cast<float>(nb)) << "ms." << std::endl;
		std::cout << "Computation time on CPU:" << (cElapsed/static_cast<float>(nb)) << "ms." << std::endl;
		std::cout << "==========================================" << std::endl;

		return nb == nbLoops;
	}

	bool test_product(const H_Matrix& hA, const H_Matrix&hB, const D_Matrix& dA, const D_Matrix&dB) {
		// do the product on GPU
		ChronoGPU gChr;
		gChr.start();
		const D_Matrix dResult = dA.product1( dB );
		gChr.stop();
		
		// ground truth (CPU)
		ChronoCPU cChr;
		cChr.start();
		H_Matrix truth = hA * hB;
		cChr.stop();

		// print the timings
		std::cout << "==========================================" << std::endl;
		std::cout << "-> Matrices product ..." << std::endl;
		std::cout << "Computation time on GPU:" << gChr.elapsedTime() << "ms." << std::endl;
		std::cout << "Computation time on CPU:" << cChr.elapsedTimeInMilliSeconds() << "ms." << std::endl;
		std::cout << "==========================================" << std::endl;

		H_Matrix hResult(hA.m_n);
		dResult.data(hResult.h_val);

		if (hA.m_n < 10) {
			std::cout << "Truth" << std::endl;
			matrix_print(truth);
			std::cout << "Device" << std::endl;
			matrix_print(hResult);
		}
		return truth == hResult;
	}

	bool test_product_opt(const H_Matrix& hA, const H_Matrix&hB, const D_Matrix& dA, const D_Matrix&dB) {
		// do the product on GPU
		ChronoGPU gChr;
		gChr.start();
		const D_Matrix dResult = dA.product2(dB);
		gChr.stop();

		// ground truth (CPU)
		ChronoCPU cChr;
		cChr.start();
		H_Matrix truth = hA * hB;
		cChr.stop();

		// print the timings
		std::cout << "==========================================" << std::endl;
		std::cout << "-> Matrices product ..." << std::endl;
		std::cout << "Computation time on GPU:" << gChr.elapsedTime() << "ms." << std::endl;
		std::cout << "Computation time on CPU:" << cChr.elapsedTimeInMilliSeconds() << "ms." << std::endl;
		std::cout << "==========================================" << std::endl;

		H_Matrix hResult(hA.m_n);
		dResult.data(hResult.h_val);

		if (hA.m_n < 10) {
			std::cout << "Truth" << std::endl;
			matrix_print(truth);
			std::cout << "Device" << std::endl;
			matrix_print(hResult);
		}
		return truth == hResult;
	}

	void doExercice_1(
		const H_Matrix&hA, const H_Matrix&hB,
		const D_Matrix&dA, const D_Matrix&dB) 
	{
		if (D_Matrix::Exo1IsDone()) {
			// check the addition
			if (test_addition(hA, hB, dA, dB))
				std::cout << "--> Well done! Your ADDITION implementation seems to work." << std::endl;
			else
				std::cout << "--> Poor luck ... Your ADDITION implementation does not work." << std::endl;
		}
		else {
			std::cout << "Exercise 1 not implemented (D_Matrix::Exo1IsDone() returns false)" << std::endl;
		}
	}

	void doExercice_2(
		const H_Matrix&hA, const H_Matrix&hB,
		const D_Matrix&dA, const D_Matrix&dB) 
	{
		if (D_Matrix::Exo2IsDone()) {
			// check the transpose
			if (test_transpose(hB, dB))
				std::cout << "--> Well done! Your TRANSPOSE implementation seems to work." << std::endl;
			else
				std::cout << "--> Poor luck ... Your TRANSPOSE implementation does not work." << std::endl;
		}
		else {
			std::cout << "Exercise 2 not implemented (D_Matrix::Exo2IsDone() returns false)" << std::endl;
		}
	}
	

	void doExercice_3(
		const H_Matrix&hA, const H_Matrix&hB,
		const D_Matrix&dA, const D_Matrix&dB) 
	{
		// check the diffusion
		if (D_Matrix::Exo3IsDone()) {
			if (test_diffusion(hA, dA))
				std::cout << "--> Well done! Your DIFFUSION implementation seems to work." << std::endl;
			else
				std::cout << "--> Poor luck ... Your DIFFUSION implementation does not work." << std::endl; 
		}
		else {
			std::cout << "Exercise 3 not implemented (D_Matrix::Exo3IsDone() returns false)" << std::endl;
		}
	}
	

	void doExercice_4(
		const H_Matrix&hA, const H_Matrix&hB,
		const D_Matrix&dA, const D_Matrix&dB) 
	{
		// check the product
		if (D_Matrix::Exo4IsDone()) {
			if (test_product(hA, hB, dA, dB))
				std::cout << "--> Well done! Your PRODUCT implementation seems to work." << std::endl;
			else
				std::cout << "--> Poor luck ... Your PRODUCT implementation does not work." << std::endl;
		}
		else {
			std::cout << "Exercise 4 not implemented (D_Matrix::Exo4IsDone() returns false)" << std::endl;
		}
	}
	

	void doExercice_5(
		const H_Matrix&hA, const H_Matrix&hB,
		const D_Matrix&dA, const D_Matrix&dB) 
	{
		// check the advanced product
		if (D_Matrix::Exo5IsDone()) {
			if (test_product_opt(hA, hB, dA, dB))
				std::cout << "--> Well done! Your efficient PRODUCT implementation seems to work." << std::endl;
			else
				std::cout << "--> Poor luck ... Your efficient PRODUCT implementation does not work." << std::endl;
		}
		else {
			std::cout << "Exercise 5 not implemented (D_Matrix::Exo5IsDone() returns false)" << std::endl;
		}			
	}
	
	void doExercises(const int N) {
		srand(static_cast<unsigned int>(time(NULL)));

		H_Matrix hA( H_Matrix::random(N) );
		H_Matrix hB( H_Matrix::random(N) ); 
		
		D_Matrix dA( hA.export2Device() );
		D_Matrix dB( hB.export2Device() );
	
		// check that equality test works
		if (!test_equality(hA)) {
			std::cerr << "test equality of two matrices does not work ... cannot proceed!" << std::endl;
			exit(EXIT_FAILURE);
		}
		else
			std::cout << "test equality with success" << std::endl;
	
		// do the 5 exercises
		doExercice_1(hA, hB, dA, dB);
		doExercice_2(hA, hB, dA, dB);
		doExercice_3(hA, hB, dA, dB);
		doExercice_4(hA, hB, dA, dB);
		doExercice_5(hA, hB, dA, dB);
	}
}

int main(int ac, const char **av) {
	std::cout << "Labwork 2 : Matrix multiplication" << std::endl;
	// find and start a device ...
	std::cout<<"Find the device ..." << std::endl;
	int bestDevice = findCudaDevice(ac, av);
	checkCudaErrors( hipSetDevice( bestDevice ) );
	
	// search parameters    
	displayHelpIfNeeded(ac, av);
	const int N = getNFromCmdLine(ac, av);
	std::cout << "Will use matrices of size " << N << "x" << N << "." << std::endl;
	std::cout << " -- a matrix needs " << (N*N*sizeof(int) >> 20) << " Mb." << std::endl;
	
	// do the exercises ...
	doExercises(N);

	return EXIT_SUCCESS;
}